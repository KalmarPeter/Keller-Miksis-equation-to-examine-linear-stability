#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include <fstream>
#define _USE_MATH_DEFINES
#include <math.h>

#define Resolution 16384
#define Res 128
#define NumberOfModes 5
#define SysDim 12
# define Perturbation 1.0e-6
#define T 1.0
#define AbsTolerance 1.0e-9
#define RelTolerance 1.0e-9
#define AbsToleranceModes 1.0e-8
#define RelToleranceModes 1.0e-8
#define ConvergedIteration 90
#define TransientIteration 512
#define GrowLimit  5.0
#define ShrinkLimit  0.1

// Functions

void Samplog(double, double, int, double*);
__global__ void GetC(double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*, double*);
void GetS(double*, double*);
__forceinline__ __device__ void OdeFun(double*, double&, double*, double*, double*, double*);
__forceinline__ __device__ void rkck(double*, double*, double*, double&, double*, double*, double*);
__forceinline__ __device__ void GetTolerance(double*, double*, double*);
__forceinline__ __device__ void GetTimeStep(double*, double*, double*, double*, double*, double&, double&, double*);
__global__ void OdeSolver(double*, double*, double*, double*, double*, double*);


clock_t SimulationStart = clock();

//Control parameters:

double R0[] = { (10.0e-6) }; // equilibrium bouble size
const int BlockSize = 32;
const int GridSize = Resolution / BlockSize + (Resolution % BlockSize == 0 ? 0 : 1);

// Features of the fluid:

double penv[] = { 1.0e5 }; // environmental pressure
double pv[] = { 3166.8 }; // vapor pressure
double n[] = { 1.4 }; // polytropic exponent
double rho[] = { 997.0 }; // density
double sft[] = { 0.072 }; // surface tension
double nu[] = { 0.00089 }; // viscosity
double cl[] = { 1497.0 }; // speed of sound in water
double theta[] = { 0.0 };

const double fr0 = (1.0 / (2.0 * M_PI * *R0 * pow(*rho, (1.0 / 2.0)))) * pow((3.0 * *n * (*penv + (2.0 * *sft / *R0) - *pv) - (2.0 * *sft / *R0) - (4.0 * pow(*nu, 2.0)) / (*rho * pow(*R0, 2.0))), (1.0 / 2.0)); // own frequency of the bubble
const double omega0 = pow(((3.0 * *n * (*penv - *pv)) / (*rho * *R0 * *R0)) + ((2.0 * (3.0 * *n - 1.0) * *sft) / (*rho * *R0)), (1.0 / 2.0)); // own angular speed of the bubble

int main()
{
	// Initial Conditions

	double* h_t0 = new double[Resolution];
	double* h_y = new double[SysDim * Resolution];
	double* h_Rm = new double[Resolution];
	double* h_l = new double[NumberOfModes];
	double* h_s = new double[5 * NumberOfModes];
	double* h_r = new double[NumberOfModes * Resolution];

	double* d_t0;
	double* d_y;
	double* d_Rm;
	double* d_s;
	double* d_r;

	hipMalloc((void**)&d_t0, Resolution * sizeof(double));
	hipMalloc((void**)&d_y, SysDim * Resolution * sizeof(double));
	hipMalloc((void**)&d_Rm, Resolution * sizeof(double));
	hipMalloc((void**)&d_s, 5 * NumberOfModes * sizeof(double));
	hipMalloc((void**)&d_r, NumberOfModes * Resolution * sizeof(double));

	for (int i = 0; i < Resolution; i++)
	{
		h_t0[i] = 0.0;
		h_y[i] = 1.0;
		h_y[i + Resolution] = 0.0;
		h_Rm[i] = *R0;
	}

	for (int i = 0; i < Resolution * NumberOfModes; i++)
	{
		h_r[i] = 0.0;
	}

	for (int x = 0; x < NumberOfModes; x++)
	{
		for (int i = 0; i < Resolution; i++)
		{
			h_y[i + (x + 1) * 2 * Resolution] = Perturbation;
			h_y[i + (2 * x + 3) * Resolution] = 0.0;
		}
	}

	h_l[0] = 2.0;
	h_l[1] = 3.0;
	h_l[2] = 4.0;
	h_l[3] = 5.0;
	h_l[4] = 6.0;

	hipMemcpy(d_t0, h_t0, Resolution * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, SysDim * Resolution * sizeof(double), hipMemcpyHostToDevice);

	// Copy fluid properties to device

	double* d_penv;
	double* d_pv;
	double* d_n;
	double* d_rho;
	double* d_sft;
	double* d_nu;
	double* d_cl;
	double* d_theta;
	double* d_R0;

	hipMalloc((void**)&d_penv, sizeof(double));
	hipMalloc((void**)&d_pv, sizeof(double));
	hipMalloc((void**)&d_n, sizeof(double));
	hipMalloc((void**)&d_rho, sizeof(double));
	hipMalloc((void**)&d_sft, sizeof(double));
	hipMalloc((void**)&d_nu, sizeof(double));
	hipMalloc((void**)&d_cl, sizeof(double));
	hipMalloc((void**)&d_theta, sizeof(double));
	hipMalloc((void**)&d_R0, sizeof(double));

	hipMemcpy(d_penv, penv, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_pv, pv, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_n, n, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_rho, rho, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_sft, sft, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_nu, nu, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_cl, cl, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_theta, theta, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_R0, R0, sizeof(double), hipMemcpyHostToDevice);

	// Parameters of the excitation

	double pa1[] = { 0.6e5 }; // pressure amplitude 1
	double pa2[] = { 0.4e5 }; // pressure amplitude 2
	double* omega1 = new double[Resolution]; // angular speed 1
	double* omega2 = new double[Resolution]; // angular speed 2
	double* fr1 = new double[Res];
	double* fr2 = new double[Res];
	double* Fr1 = new double[Resolution];
	double* Fr2 = new double[Resolution];

	// Logaritmic sampling

	double LowerBoundary = 20.0e3;
	double UpperBoundary = 2.0e6;
	Samplog(LowerBoundary, UpperBoundary, Res, fr1);
	Samplog(LowerBoundary, UpperBoundary, Res, fr2);

	// Copy the exacition parameters to the device

	for (int i = 0; i < Res; i++)
	{
		for (int j = 0; j < Res; j++)
		{
			Fr1[j + i * Res] = fr1[j];
			Fr2[j + i * Res] = fr2[i];
			omega1[j + i * Res] = 2.0 * M_PI * fr1[j];
			omega2[j + i * Res] = 2.0 * M_PI * fr2[i];
		}
	}

	double* d_pa1;
	double* d_pa2;
	double* d_omega1;
	double* d_omega2;

	hipMalloc((void**)&d_pa1, sizeof(double));
	hipMalloc((void**)&d_pa2, sizeof(double));
	hipMalloc((void**)&d_omega1, Resolution * sizeof(double));
	hipMalloc((void**)&d_omega2, Resolution * sizeof(double));

	hipMemcpy(d_pa1, pa1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_pa2, pa2, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_omega1, omega1, Resolution * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_omega2, omega2, Resolution * sizeof(double), hipMemcpyHostToDevice);

	// Create a txt

	std::ofstream DataFile;
	DataFile.open("rkck_cuda.txt");
	int Width = 18;
	DataFile.precision(10);
	DataFile.flags(std::ios::scientific);

	// Define d_c pointer for GetC function

	double* d_c;
	hipMalloc((void**)&d_c, 17 * Resolution * sizeof(double));

	// Get constans for OdeFun

	GetC << <GridSize, BlockSize >> > (d_omega1, d_omega2, d_c, d_penv, d_pv, d_R0, d_rho, d_cl, d_n, d_sft, d_pa1, d_pa2, d_nu, d_theta);

	GetS(h_s, h_l);

	// Ode Solver

	hipMemcpy(d_s, h_s, 5 * NumberOfModes * sizeof(double), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	for (int z = 0; z < TransientIteration; z++)
	{
		OdeSolver << <GridSize, BlockSize >> > (d_t0, d_y, d_c, d_s, d_Rm, d_r);
		hipDeviceSynchronize();
		std::cout << z << " t" << std::endl;
	}


	hipMemcpy(d_Rm, h_Rm, sizeof(double) * Resolution, hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_r, sizeof(double) * NumberOfModes * Resolution, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	for (int z = 0; z < ConvergedIteration; z++)
	{
		OdeSolver << <GridSize, BlockSize >> > (d_t0, d_y, d_c, d_s, d_Rm, d_r);
		hipDeviceSynchronize();
		std::cout << z << " c" << std::endl;
	}

	hipMemcpy(h_r, d_r, Resolution * NumberOfModes * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_Rm, d_Rm, Resolution * sizeof(double), hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	//Write maximum bubble radius at the given sampleing to txt

	for (int j = 0; j < NumberOfModes; j++)
	{
		for (int i = 0; i < Resolution; ++i)
		{
			DataFile.width(Width); DataFile << (h_r[i + j * Resolution] / ConvergedIteration) * Fr1[i] / fr0 << ',';
			DataFile.width(Width); DataFile << Fr1[i] << ',';
			DataFile.width(Width); DataFile << h_Rm[i] << ',';
			DataFile.width(Width); DataFile << Fr2[i];
			DataFile << '\n';
		}
	}

	DataFile.close();

	// Delete pointers

	hipFree(d_omega1);
	hipFree(d_omega2);
	hipFree(d_c);
	hipFree(d_penv);
	hipFree(d_pv);
	hipFree(d_R0);
	hipFree(d_rho);
	hipFree(d_cl);
	hipFree(d_n);
	hipFree(d_sft);
	hipFree(d_pa1);
	hipFree(d_pa2);
	hipFree(d_nu);
	hipFree(d_theta);
	hipFree(d_t0);
	hipFree(d_y);
	hipFree(d_Rm);
	hipFree(d_s);
	hipFree(d_r);

	//delete[] omega1, omega2, pa1,pa2,h_t0,h_y;
	//delete[] penv, pv, R0, rho, cl, n, sft, nu, theta;

	clock_t SimulationEnd = clock();
	std::cout << 1000 * (SimulationEnd - SimulationStart) / CLOCKS_PER_SEC << std::endl;

	return 0;
}

// Logaritmic sampling

void Samplog(double LowerBoundary, double UpperBoundary, int NumberOfIntervals, double* Intervals)
{
	double LogUpper = log10(UpperBoundary);
	double LogLower = log10(LowerBoundary);
	double Steps = (LogUpper - LogLower) / (NumberOfIntervals - 1.0);

	Intervals[0] = LowerBoundary;
	Intervals[NumberOfIntervals - 1] = UpperBoundary;

	for (int i = 1; i < NumberOfIntervals - 1; i++)
	{
		Intervals[i] = pow(10, LogLower + i * Steps);
	}
}

// Constans for dimensionless Ryleigh Plesset equation

__global__ void GetC(double* d_omega1, double* d_omega2, double* d_c, double* d_penv, double* d_pv, double* d_R0, double* d_rho, double* d_cl, double* d_n, double* d_sft, double* d_pa1, double* d_pa2, double* d_nu, double* d_theta)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	double r_pa1 = *d_pa1;
	double r_pa2 = *d_pa2;
	double r_penv = *d_penv;
	double r_pv = *d_pv;
	double r_R0 = *d_R0;
	double r_rho = *d_rho;
	double r_cl = *d_cl;
	double r_n = *d_n;
	double r_sft = *d_sft;
	double r_nu = *d_nu;
	double r_theta = *d_theta;

	d_c[tid] = (((4.0 * M_PI * M_PI) / (r_R0 * r_R0 * d_omega1[tid] * d_omega1[tid] * r_rho)) * (((2.0 * r_sft) / (r_R0)) + r_penv - r_pv));
	d_c[tid + Resolution] = ((1.0 - 3.0 * r_n) * (2 * r_sft / r_R0 + r_penv - r_pv) * (2.0 * M_PI / r_R0 / d_omega1[tid]) / r_cl / r_rho);
	d_c[tid + 2 * Resolution] = (((r_penv - r_pv) * 4.0 * M_PI * M_PI) / (r_R0 * r_R0 * d_omega1[tid] * d_omega1[tid] * r_rho));
	d_c[tid + 3 * Resolution] = ((8.0 * M_PI * M_PI * r_sft) / (r_R0 * r_R0 * r_R0 * d_omega1[tid] * d_omega1[tid] * r_rho));
	d_c[tid + 4 * Resolution] = ((8.0 * M_PI * r_nu) / (r_R0 * r_R0 * d_omega1[tid] * r_rho));
	d_c[tid + 5 * Resolution] = ((4.0 * M_PI * M_PI * r_pa1) / (r_R0 * r_R0 * d_omega1[tid] * d_omega1[tid] * r_rho));
	d_c[tid + 6 * Resolution] = ((4.0 * M_PI * M_PI * r_pa2) / (r_R0 * r_R0 * d_omega1[tid] * d_omega1[tid] * r_rho));
	d_c[tid + 7 * Resolution] = ((4.0 * M_PI * M_PI * r_pa1) / (r_R0 * d_omega1[tid] * r_rho * r_cl));
	d_c[tid + 8 * Resolution] = ((4.0 * M_PI * M_PI * d_omega2[tid] * r_pa2) / (r_R0 * d_omega1[tid] * d_omega1[tid] * r_rho * r_cl));
	d_c[tid + 9 * Resolution] = ((r_R0 * d_omega1[tid]) / (2.0 * M_PI * r_cl));
	d_c[tid + 10 * Resolution] = (3.0 * r_n);
	d_c[tid + 11 * Resolution] = ((d_omega2[tid]) / (d_omega1[tid]));
	d_c[tid + 12 * Resolution] = (r_theta);
	d_c[tid + 13 * Resolution] = (2.0 * M_PI / d_omega1[tid]);
	d_c[tid + 14 * Resolution] = r_R0;

	if (r_pa1 != 0.0)
	{
		d_c[tid + 15 * Resolution] = (sqrt(r_nu / r_rho / d_omega1[tid]));
	}
	else
	{
		d_c[tid + 15 * Resolution] = 0.0;
	}

	if (r_pa2 != 0.0)
	{
		d_c[tid + 16 * Resolution] = (sqrt(r_nu / r_rho / d_omega2[tid]));
	}
	else
	{
		d_c[tid + 16 * Resolution] = 0.0;
	}

}

void GetS(double* h_s, double* h_l)
{
	for (int i = 0; i < NumberOfModes; i++)
	{
		h_s[5 * i] = h_l[i];
		h_s[5 * i + 1] = h_l[i] - 1.0;
		h_s[5 * i + 2] = h_l[i] * (h_l[i] + 2.0) * (h_l[i] + 2.0);
		h_s[5 * i + 3] = h_l[i] * (h_l[i] - 1.0) * (h_l[i] + 2.0);
		h_s[5 * i + 4] = (h_l[i] - 1.0) * (h_l[i] + 1.0) * (h_l[i] + 2.0);
	}
}

// Dimensionless Keller - Miksis equation

__forceinline__ __device__ void OdeFun(double* f, double& dt, double* t0, double* y, double* c, double* d_s)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	double rply;
	double N;
	double D;
	double arg1;
	double arg2;
	double NPerD;
	rply = 1.0 / y[tid];
	arg1 = 2.0 * M_PI * t0[tid];
	arg2 = 2.0 * M_PI * c[tid + 11 * Resolution] * t0[tid] + c[tid + 12 * Resolution];

	f[0] = dt * y[tid + Resolution];

	N = (c[tid] + c[tid + Resolution] * y[tid + Resolution]) * pow(rply, c[tid + 10 * Resolution]) - c[tid + 2 * Resolution] * (1.0 + c[tid + 9 * Resolution] * y[tid + Resolution]) - c[tid + 3 * Resolution] * rply - c[tid + 4 * Resolution] * y[tid + Resolution] * rply
		- (1.5 - 0.5 * c[tid + 9 * Resolution] * y[tid + Resolution]) * y[tid + Resolution] * y[tid + Resolution] - (c[tid + 5 * Resolution] * sin(arg1) + c[tid + 6 * Resolution] * sin(arg2)) * (1.0 + c[tid + 9 * Resolution] * y[tid + Resolution])
		- y[tid] * (c[tid + 7 * Resolution] * cos(arg1) + c[tid + 8 * Resolution] * cos(arg2));

	D = y[tid] - c[tid + 9 * Resolution] * y[tid] * y[tid + Resolution] + c[tid + 4 * Resolution] * c[tid + 9 * Resolution];

	NPerD = N / D;

	f[1] = dt * NPerD;

	// Modes

	int t = threadIdx.x;

	__shared__ double s_s[5 * NumberOfModes];

	if (t < 5 * NumberOfModes)
	{
		s_s[t] = d_s[t];
	}
	__syncthreads();

	double blt[NumberOfModes]; // boundary layer thickness
	double A[NumberOfModes];
	double B[NumberOfModes];
	double E[NumberOfModes];
	double yc14;


	for (int i = 0; i < NumberOfModes; i++)
	{
		yc14 = (y[tid] * c[tid + 14 * Resolution]);
		blt[i] = 2.0 * (fmin((fmax(c[tid + 15 * Resolution], c[tid + 16 * Resolution])), yc14 / (2.0 * s_s[5 * i])));
		E[i] = (1.0 / (1.0 + blt[i] / yc14));
	}

	for (int i = 0; i < NumberOfModes; i++)
	{
		A[i] = (-s_s[5 * i + 1] * NPerD * rply + s_s[5 * i + 4] * c[tid + 3 * Resolution] * 0.5 * rply * rply * rply + 0.5 * c[tid + 4 * Resolution] * (s_s[5 * i + 4] - s_s[5 * i + 3] * E[i]) * y[tid + Resolution] * rply * rply * rply);

		B[i] = (3.0 * y[tid + Resolution] * rply + 0.5 * c[tid + 4 * Resolution] * rply * rply * (s_s[5 * i + 2] * E[i] - s_s[5 * i + 4]));
	}

	for (int i = 0; i < NumberOfModes; i++)
	{
		f[2 * i + 2] = dt * (y[tid + Resolution * (2 * i + 3)]);

		f[2 * i + 3] = dt * (-A[i] * y[tid + 2 * Resolution * (i + 1)] - B[i] * y[tid + Resolution * (2 * i + 3)]);
	}
}

// Runge-Kutta-Cash-Karp Method

__forceinline__ __device__ void rkck(double* t0, double* y, double* c, double& dt, double* yn, double* error, double* d_s)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	double k1[SysDim];
	double k2[SysDim];
	double k3[SysDim];
	double k4[SysDim];
	double k5[SysDim];
	double k6[SysDim];
	double yact[SysDim];


	//k1

	OdeFun(k1, dt, t0, y, c, d_s);

	//k2

	double hk2[1];
	hk2[0] = t0[tid] + (1.0 / 5.0) * dt;

	for (int i = 0; i < SysDim; ++i)
	{
		yact[tid + i * Resolution] = y[tid + i * Resolution] + (1.0 / 5.0) * k1[i];
	}

	OdeFun(k2, dt, hk2, yact, c, d_s);

	//k3

	double hk3[1];
	hk3[0] = t0[tid] + (3.0 / 10.0) * dt;

	for (int i = 0; i < SysDim; ++i)
	{
		yact[tid + i * Resolution] = y[tid + i * Resolution] + (3.0 / 40.0) * k1[i] + (9.0 / 40.0) * k2[i];
	}

	OdeFun(k3, dt, hk3, yact, c, d_s);

	//k4

	double hk4[1];
	hk4[0] = t0[tid] + (3.0 / 5.0) * dt;

	for (int i = 0; i < SysDim; ++i)
	{
		yact[tid + i * Resolution] = y[tid + i * Resolution] + (3.0 / 10.0) * k1[i] + (-9.0 / 10.0) * k2[i] + (6.0 / 5.0) * k3[i];
	}

	OdeFun(k4, dt, hk4, yact, c, d_s);

	//k5

	double hk5[1];
	hk5[0] = t0[tid] + (1.0) * dt;

	for (int i = 0; i < SysDim; ++i)
	{
		yact[tid + i * Resolution] = y[tid + i * Resolution] + (-11.0 / 54.0) * k1[i] + (5.0 / 2.0) * k2[i] + (-70.0 / 27.0) * k3[i] + (35.0 / 27.0) * k4[i];
	}

	OdeFun(k5, dt, hk5, yact, c, d_s);

	//k6

	double hk6[1];
	hk6[0] = t0[tid] + (7.0 / 8.0) * dt;

	for (int i = 0; i < SysDim; ++i)
	{
		yact[tid + i * Resolution] = y[tid + i * Resolution] + (1631.0 / 55296.0) * k1[i] + (175.0 / 512.0) * k2[i] + (575.0 / 13824.0) * k3[i] + (44275.0 / 110592.0) * k4[i] + (253.0 / 4096.0) * k5[i];
	}

	OdeFun(k6, dt, hk6, yact, c, d_s);

	// yn

	for (int i = 0; i < SysDim; ++i)
	{
		yn[tid + i * Resolution] = y[tid + i * Resolution] + (37.0 / 378.0) * k1[i] + (0.0) * k2[i] + (250.0 / 621.0) * k3[i] + (125.0 / 594.0) * k4[i] + (0.0) * k5[i] + (512.0 / 1771.0) * k6[i];
	}

	// error

	for (int i = 0; i < SysDim; ++i)
	{
		error[tid + i * Resolution] = fabs(((37.0 / 378.0) - (2825.0 / 27648.0)) * k1[i] + (0.0) * k2[i] + ((250.0 / 621.0) - (18575.0 / 48384.0)) * k3[i] + ((125.0 / 594.0) - (13525.0 / 55296.0)) * k4[i] + ((0.0) - (277.0 / 14336.0)) * k5[i] + ((512.0 / 1771.0) - (1.0 / 4.0)) * k6[i]) + 1.0e-30;
	}
}

//Define tolerances

__forceinline__ __device__ void GetTolerance(double* y, double* yn, double* tol)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	double abstol[SysDim];
	double reltol[SysDim];
	double reltolacty[SysDim];
	double reltolactyn[SysDim];
	double reltolact[SysDim];
	double habs = 1.0e-300;
	double hrelact = 1.0e-300;

	for (int i = 0; i < 2; i++)
	{
		abstol[i] = AbsTolerance;
		reltol[i] = RelTolerance;
	}

	for (int i = 2; i < SysDim; i++)
	{
		abstol[i] = AbsToleranceModes;
		reltol[i] = RelToleranceModes;
	}

	for (int x = 0; x < SysDim; ++x)
	{
		reltolacty[x] = reltol[x] * fabs(y[tid + x * Resolution]);
		reltolactyn[x] = reltol[x] * fabs(yn[tid + x * Resolution]);
	}

	for (int x = 0; x < SysDim; x++)
	{
		reltolact[x] = fmin(reltolacty[x], reltolactyn[x]);
	}


	for (int i = 0; i < SysDim; ++i)
	{
		if (abstol[i] > habs)
		{
			habs = abstol[i];
		}

		if (reltolact[i] > hrelact)
		{
			hrelact = reltolact[i];
		}
	}

	if (habs >= hrelact)
	{
		for (int x = 0; x < SysDim; ++x)
		{
			tol[tid + x * Resolution] = abstol[x];
		}
	}

	else
	{
		for (int x = 0; x < SysDim; ++x)
		{
			tol[tid + x * Resolution] = reltolact[x];
		}
	}
}

//Calculate the following time step

__forceinline__ __device__ void GetTimeStep(double* tol, double* error, double* y, double* yn, double* t0, double& dt, double& t, double* Rm)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	double MintolDivError = 1.0e300;
	double TimeStepper;
	double MaxTimeStep = 1.0e6;
	double MinTimeStep = 1.0e-12;
	bool Update = false;

	for (int i = 0; i < SysDim; ++i)
	{
		if ((tol[tid + i * Resolution] / error[tid + i * Resolution]) < MintolDivError)
		{
			MintolDivError = (tol[tid + i * Resolution] / error[tid + i * Resolution]);
		}
	}

	if (MintolDivError >= 1)
	{
		Update = 1;
	}

	if (Update == 1)
	{
		TimeStepper = 0.9 * pow(MintolDivError, 0.2);
	}

	else
	{
		TimeStepper = 0.9 * pow(MintolDivError, 0.25);
	}

	if (isfinite(TimeStepper) == 0)
	{
		Update = 0;
	}

	if (Update == 1)
	{
		for (int x = 0; x < SysDim; ++x)
		{
			y[tid + x * Resolution] = yn[tid + x * Resolution];
		}

		t0[tid] += dt;
		t += dt;
	}

	TimeStepper = fmin(TimeStepper, GrowLimit);
	TimeStepper = fmax(TimeStepper, ShrinkLimit);

	dt = dt * TimeStepper;

	dt = fmin(dt, MaxTimeStep);
	dt = fmax(dt, MinTimeStep);

	if ((t + dt) > T)
	{
		dt = T - t;
	}

	if (y[tid] > Rm[tid])
	{
		Rm[tid] = y[tid];
	}
}

// ODE Solver

__global__ void OdeSolver(double* d_t0, double* d_y, double* c, double* d_s, double* d_Rm, double* d_r)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	double r_t = 0.0;
	double r_dt = 3.016e-6;
	double r_yn[SysDim];
	double r_error[SysDim];
	double r_tol[SysDim];
	double r_r[NumberOfModes];

	while (r_t < T)
	{
		rkck(d_t0, d_y, c, r_dt, r_yn, r_error, d_s);
		GetTolerance(d_y, r_yn, r_tol);
		GetTimeStep(r_tol, r_error, d_y, r_yn, d_t0, r_dt, r_t, d_Rm);
	}

	// Define initial perturbation for modes, caculate grow rate

	for (int i = 0; i < NumberOfModes; i++)
	{
		r_r[i] = fabs(d_y[tid + (i + 1) * 2 * Resolution] / Perturbation);

		if (r_r[i] > 0.0)
		{
			d_y[tid + 2 * (i + 1) * Resolution] /= r_r[i];
			d_y[tid + (2 * i + 3) * Resolution] /= r_r[i];

			d_r[tid + i * Resolution] += log(r_r[i]);
		}

	}
}